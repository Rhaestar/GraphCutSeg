#include "hip/hip_runtime.h"
#include "gpuimplem.hh"

#include <cstdint>
#include <iostream>
#include <cstdlib>
#include <cfloat>
#include "SDL.h"

namespace GPU
{
    template<typename T>
    __device__ inline T* eltPtr(T *baseAddress, size_t col, size_t row,
        size_t pitch)
    {
        return (T*)((char*)baseAddress + row * pitch + col * sizeof(T));
    }

    void fillHists(Histogram& foreHist, Histogram& backHist, SDL_Surface* image,
        SDL_Surface* mask, uint8_t* bitmask)
    {
        SDL_LockSurface(image);
        SDL_LockSurface(mask);

        uint8_t* pixels = (uint8_t*)image->pixels;
        uint8_t* masks  = (uint8_t*)mask->pixels;

        uint8_t r;
        uint8_t g;
        uint8_t b;

        SDL_PixelFormat* fmti = image->format;
        SDL_PixelFormat* fmtm = mask->format;

        for (int i = 0; i < image->h; ++i)
        {
            for (int j = 0; j < image->w; ++j)
            {
                uint32_t pixelm = *(uint32_t*)(masks + i * mask->pitch +
                    j * 4);
                SDL_GetRGB(pixelm, fmtm, &r, &g, &b);
                if (r == 255 && g == 0 && b == 0)
                {
                    uint32_t pixeli = *(uint32_t*)(pixels + i * image->pitch +
                        j * 4);
                    SDL_GetRGB(pixeli, fmti, &r, &g, &b);
                    foreHist.AddElement(r, g, b);
                    bitmask[i * image->w + j] = 1;
                }

                else if (r == 0 && g == 0 && b == 255)
                {
                    uint32_t pixeli = *(uint32_t*)(pixels + i * image->pitch +
                        j * 4);
                    SDL_GetRGB(pixeli, fmti, &r, &g, &b);
                    backHist.AddElement(r, g, b);
                    bitmask[i * image->w + j] = 2;
                }
            }
        }

        SDL_UnlockSurface(image);
        SDL_UnlockSurface(mask);
    }

    int ComputeDiff(uint8_t r1, uint8_t g1, uint8_t b1,
        uint8_t r2, uint8_t g2, uint8_t b2, float sigma, int param)
    {
        int rdiff = r1 - r2;
        int gdiff = g1 - g2;
        int bdiff = b1 - b2;

        float sqdiff = rdiff * rdiff + gdiff * gdiff + bdiff * bdiff;
        return (int)((float)(param) * expf(-sqdiff / (2 * sigma * sigma)));
    }

    int InitializeCapacities(int* weightsUp, int* weightsDown,
        int* weightsLeft, int* weightsRight, SDL_Surface* image,
        float sigma, int param)
    {
        SDL_LockSurface(image);

        int maxCap = 0;

        uint8_t* pixels = (uint8_t*)image->pixels;
        SDL_PixelFormat* fmt = image->format;

        uint8_t r1;
        uint8_t g1;
        uint8_t b1;
        uint8_t r2;
        uint8_t g2;
        uint8_t b2;


        for (int i = 0; i < image->h; ++i)
        {
            for (int j = 0; j < image->w; ++j)
            {
                uint32_t pixel = *(uint32_t*)(pixels + i * image->pitch +
                    j * 4);
                SDL_GetRGB(pixel, fmt, &r1, &g1, &b1);

                if (i > 0)
                {
                    uint32_t pixel2 = *(uint32_t*)(pixels + (i - 1) *
                        image->pitch + j * 4);
                    SDL_GetRGB(pixel2, fmt, &r2, &g2, &b2);
                    weightsUp[i * image->w + j] =
                        ComputeDiff(r1, g1, b1, r2, g2, b2, sigma, param);
                    maxCap = std::max(maxCap, weightsUp[i * image->w + j]);
                }
                if (i < image->h - 1)
                {
                    uint32_t pixel2 = *(uint32_t*)(pixels + (i + 1) *
                        image->pitch + j * 4);
                    SDL_GetRGB(pixel2, fmt, &r2, &g2, &b2);
                    weightsDown[i * image->w + j] =
                        ComputeDiff(r1, g1, b1, r2, g2, b2, sigma, param);
                    maxCap = std::max(maxCap, weightsDown[i * image->w + j]);
                }
                if (j > 0)
                {
                    uint32_t pixel2 = *(uint32_t*)(pixels + i *
                        image->pitch + (j - 1) * 4);
                    SDL_GetRGB(pixel2, fmt, &r2, &g2, &b2);
                    weightsLeft[i * image->w + j] =
                        ComputeDiff(r1, g1, b1, r2, g2, b2, sigma, param);
                    maxCap = std::max(maxCap, weightsLeft[i * image->w + j]);
                }
                if (j < image->w - 1)
                {
                    uint32_t pixel2 = *(uint32_t*)(pixels + i *
                        image->pitch + (j + 1) * 4);
                    SDL_GetRGB(pixel2, fmt, &r2, &g2, &b2);
                    weightsRight[i * image->w + j] =
                        ComputeDiff(r1, g1, b1, r2, g2, b2, sigma, param);
                    maxCap = std::max(maxCap, weightsRight[i * image->w + j]);
                }
            }
        }

        SDL_UnlockSurface(image);

        return maxCap;
    }

    void InitializeExcess(int *excessFlows, SDL_Surface* image,
        Histogram& foreHist, Histogram& backHist,
        uint8_t* bitmask, int k, float lambda)
    {
        SDL_LockSurface(image);

        uint8_t* pixels = (uint8_t*)image->pixels;
        SDL_PixelFormat* fmt = image->format;

        uint8_t r;
        uint8_t g;
        uint8_t b;

        uint32_t pixel;

        float pobj;
        float pbkg;

        for (int i = 0; i < image->h; ++i)
        {
            for (int j = 0; j < image->w; ++j)
            {
                switch (bitmask[i * image->w + j])
                {
                    case 0:
                        pixel = *(uint32_t*)(pixels + i *
                            image->pitch + j * 4);
                        SDL_GetRGB(pixel, fmt, &r, &g, &b);
                        pobj = foreHist.GetProba(r, g, b);
                        pbkg = backHist.GetProba(r, g, b);
                        if (pobj == 0.f)
                            pobj = FLT_MIN;
                        if (pbkg == 0.f)
                            pbkg = FLT_MIN;
                        pobj = -logf(pobj);
                        pbkg = -logf(pbkg);
                        excessFlows[i * image->w + j] = (int)(lambda * pbkg -
                            lambda * pobj);
                        break;
                    case 1:
                        excessFlows[i * image->w + j] = k;
                        break;
                    case 2:
                        excessFlows[i * image->w + j] = -k;
                        break;
                    default:
                        std::cout << "default\n";
                        break;
                }
            }
        }

        SDL_UnlockSurface(image);

    }

    int IsAnyActive(int* excessFlows, uint32_t* heights, uint32_t width,
        uint32_t height, uint32_t heightMax)
    {
        int ret = 0;
        for (uint32_t i = 0; i < height; ++i)
        {
            for (uint32_t j = 0; j < width; ++j)
            {
                bool test1 = excessFlows[i * width + j] > 0;
                int it = excessFlows[i * width + j];
                it += 1;
                bool test2 = heights[i * width + j] < heightMax;
                if (test1 && test2)
                    ret++;
            }
        }
        return ret;
    }

    __global__ void Push(int* excessFlows, int* weightsUp, int* weightsDown,
        int* weightsLeft, int* weightsRight, uint32_t* heights,
        uint32_t heightMax, uint32_t width, uint32_t height, size_t pitch, 
        int* isAnyActive)
    {
        size_t x = blockDim.x * blockIdx.x + threadIdx.x;
        size_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        int* currFlow = eltPtr(excessFlows, x, y, pitch);
        uint32_t currHeight = *eltPtr(heights, x, y, pitch);

        if (*currFlow > 0 && currHeight < heightMax)
        {
            if (y > 0 && currHeight - 1 == *eltPtr(heights, x, y - 1, pitch))
            {
                int *wUp = eltPtr(weightsUp, x, y, pitch);
                int *wDown = eltPtr(weightsUp, x, y - 1, pitch);
                int *currFlowEx = eltPtr(excessFlows, x, y - 1, pitch);
                int flow = min(*currFlow, *wUp);
                atomicSub(currFlow, flow);
                atomicSub(wUp, flow);
                atomicAdd(wDown, flow);
                atomicAdd(currFlowEx, flow);
            }
            if (x > 0 && currHeight - 1 == *eltPtr(heights, x - 1, y, pitch))
            {
                int *wLeft = eltPtr(weightsUp, x, y, pitch);
                int *wRight = eltPtr(weightsUp, x - 1, y, pitch);
                int *currFlowEx = eltPtr(excessFlows, x - 1, y, pitch);
                int flow = min(*currFlow, *wLeft);
                atomicSub(currFlow, flow);
                atomicSub(wLeft, flow);
                atomicAdd(wRight, flow);
                atomicAdd(currFlowEx, flow);
            }
            if (y < height - 1 && currHeight - 1 == *eltPtr(heights, x, y + 1,
                pitch))
            {
                int *wDown = eltPtr(weightsUp, x, y, pitch);
                int *wUp = eltPtr(weightsUp, x, y + 1, pitch);
                int *currFlowEx = eltPtr(excessFlows, x, y + 1, pitch);
                int flow = min(*currFlow, *wUp);
                atomicSub(currFlow, flow);
                atomicSub(wDown, flow);
                atomicAdd(wUp, flow);
                atomicAdd(currFlowEx, flow);
            }
            if (x < width - 1 && currHeight - 1 == *eltPtr(heights, x + 1, y,
                pitch))
            {
                int *wRight = eltPtr(weightsUp, x, y, pitch);
                int *wLeft = eltPtr(weightsUp, x + 1, y, pitch);
                int *currFlowEx = eltPtr(excessFlows, x + 1, y, pitch);
                int flow = min(*currFlow, *wRight);
                atomicSub(currFlow, flow);
                atomicSub(wRight, flow);
                atomicAdd(wLeft, flow);
                atomicAdd(currFlowEx, flow);
            }

            if (*currFlow > 0)
                atomicAdd(isAnyActive,1);
        }
    }

    __global__ void Relabel(int* excessFlows,
        int* weightsUp, int* weightsDown, int* weightsLeft,
        int* weightsRight, uint32_t* heights, uint32_t* heightsTemp,
        uint32_t heightMax, unsigned width, unsigned height, size_t pitch)
    {
        size_t x = blockDim.x * blockIdx.x + threadIdx.x;
        size_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if (x >= width || y >= height)
            return;

        int* currFlow = eltPtr(excessFlows, x, y, pitch);
        uint32_t* currHeight = eltPtr(heights, x, y, pitch);
        uint32_t* currHeightTemp = eltPtr(heightsTemp, x, y, pitch);

        if (*currFlow > 0 && *currHeight < heightMax)
        {
            uint32_t newHeight = heightMax;
            if (y > 0 && *eltPtr(weightsUp, x, y, pitch) > 0)
            {
                newHeight = min(newHeight, *eltPtr(heights, x, y - 1,
                    pitch) + 1);
            }
            if (x > 0 && *eltPtr(weightsLeft, x, y, pitch) > 0)
            {
                newHeight = min(newHeight, *eltPtr(heights, x - 1, y,
                    pitch) + 1);
            }
            if (y < height - 1 && *eltPtr(weightsDown, x, y, pitch) > 0)
            {
                newHeight = min(newHeight, *eltPtr(heights, x, y + 1,
                    pitch) + 1);
            }
            if (x < width - 1 && *eltPtr(weightsRight, x, y, pitch) > 0)
            {
                newHeight = min(newHeight, *eltPtr(heights, x + 1, y,
                    pitch) + 1);
            }

            *currHeightTemp = newHeight;
        }
    }

    void SavePicture(uint32_t* heights, uint32_t width, uint32_t height,
        uint32_t maxHeight)
    {
        SDL_Surface *image;

        image = SDL_CreateRGBSurface(0, width, height, 32,0,0,0,0);

        SDL_LockSurface(image);

        uint8_t* pixels = (uint8_t*)image->pixels;
        SDL_PixelFormat* fmt = image->format;

        for(unsigned i = 0; i < height; ++i)
        {
            for (unsigned j = 0; j < width; ++j)
            {
                uint32_t* pixel = (uint32_t*)(pixels + i * image->pitch +
                    j * 4);
                if (heights[i * width + j] > maxHeight - maxHeight / 3.f)
                    *pixel = SDL_MapRGBA(fmt, 255, 255, 255, 255);
                else
                    *pixel = SDL_MapRGBA(fmt, 0, 0, 0, 255);

            }
        }

        SDL_UnlockSurface(image);
        SDL_SaveBMP(image, "output.bmp");
    }

    void Implem(SDL_Surface* image, SDL_Surface* mask)
    {
        Histogram backHist;
        Histogram foreHist;

        uint32_t width = image->w;
        uint32_t height = image->h;
        uint32_t heightMax = 100;
        float sigma = 10.f;
        float lambda = 1.f;
        int param = 1;

        uint8_t* bitmask = (uint8_t*)calloc(height * width, sizeof(uint8_t));

        int* weightsUp    = (int*)calloc(height * width, sizeof(int));
        int* weightsDown  = (int*)calloc(height * width, sizeof(int));
        int* weightsLeft  = (int*)calloc(height * width, sizeof(int));
        int* weightsRight = (int*)calloc(height * width, sizeof(int));

        uint32_t* heights = (uint32_t*)calloc(height * width,
            sizeof(uint32_t));

        int* excessFlows = (int*)calloc(height * width, sizeof(int));

        fillHists(foreHist, backHist, image, mask, bitmask);

        int maxCap = InitializeCapacities(weightsUp, weightsDown,
            weightsLeft, weightsRight, image, sigma, param);

        maxCap += 1;

        InitializeExcess(excessFlows, image, foreHist, backHist, bitmask,
            maxCap, lambda);

        int block_size = 32;
        int w = std::ceil((float)width / block_size);
        int h = std::ceil((float)width / block_size);

        dim3 dimBlock(block_size, block_size);
        dim3 dimGrid(w, h);

        size_t pitch;
        int *d_weightsUp, *d_weightsDown, *d_weightsLeft, *d_weightsRight,
            *d_excessFlows;

        uint32_t *d_heights, *d_heightsTemp;

        hipMallocPitch((void **) &d_weightsUp, &pitch, width * sizeof(int),
            height);
        hipMallocPitch((void **) &d_weightsDown, &pitch, width * sizeof(int),
            height);
        hipMallocPitch((void **) &d_weightsLeft, &pitch, width * sizeof(int),
            height);
        hipMallocPitch((void **) &d_weightsRight, &pitch, width * sizeof(int),
            height);
        hipMallocPitch((void **) &d_excessFlows, &pitch, width * sizeof(int),
            height);

        hipMallocPitch((void **) &d_heights, &pitch, width * sizeof(int),
            height);
        hipMallocPitch((void **) &d_heightsTemp, &pitch, width * sizeof(int),
            height);

        hipMemcpy2D(d_weightsUp, pitch, weightsUp, width * sizeof(int),
            width * sizeof(int), height, hipMemcpyHostToDevice);
        hipMemcpy2D(d_weightsDown, pitch, weightsDown, width * sizeof(int),
            width * sizeof(int), height, hipMemcpyHostToDevice);
        hipMemcpy2D(d_weightsLeft, pitch, weightsLeft, width * sizeof(int),
            width * sizeof(int), height, hipMemcpyHostToDevice);
        hipMemcpy2D(d_weightsRight, pitch, weightsRight, width * sizeof(int),
            width * sizeof(int), height, hipMemcpyHostToDevice);
        hipMemcpy2D(d_excessFlows, pitch, excessFlows, width * sizeof(int),
            width * sizeof(int), height, hipMemcpyHostToDevice);

        hipMemcpy2D(d_heights, pitch, heights, width * sizeof(uint32_t),
            width * sizeof(uint32_t), height, hipMemcpyHostToDevice);
        hipMemcpy2D(d_heightsTemp, pitch, d_heights, pitch,
            width * sizeof(uint32_t), height, hipMemcpyDeviceToDevice);

        unsigned ip = 0;

        int isAnyActive = 1;
        int falseUtil = 0;
        int *d_isAnyActive;
        hipMalloc((void**) &d_isAnyActive, sizeof(int));

        while (ip < 1000 && isAnyActive != 0)
        {
            hipMemcpy(d_isAnyActive, &falseUtil, sizeof(int),
                hipMemcpyHostToDevice);

            Relabel<<<dimGrid, dimBlock>>>(d_excessFlows,
                d_weightsUp, d_weightsDown, d_weightsLeft, d_weightsRight,
                d_heights, d_heightsTemp, heightMax,
                width, height, pitch);
            
            hipDeviceSynchronize();

            hipMemcpy2D(d_heights, pitch, d_heightsTemp, pitch,
                width * sizeof(uint32_t), height, hipMemcpyDeviceToDevice);

            hipDeviceSynchronize();

            Push<<<dimGrid, dimBlock>>>(d_excessFlows,
                d_weightsUp, d_weightsDown, d_weightsLeft, d_weightsRight,
                d_heights, heightMax, width, height, pitch, d_isAnyActive);

            hipDeviceSynchronize();

            hipMemcpy(&isAnyActive, d_isAnyActive, sizeof(int),
                hipMemcpyDeviceToHost);
            ip++;
        }

        hipMemcpy2D(heights, width * sizeof(uint32_t), d_heights, pitch,
            width * sizeof(uint32_t), height, hipMemcpyDeviceToHost);
        SavePicture(heights, width, height, heightMax);

        hipFree(d_weightsUp);
        hipFree(d_weightsDown);
        hipFree(d_weightsLeft);
        hipFree(d_weightsRight);
        hipFree(d_excessFlows);
        hipFree(d_heights);
        hipFree(d_heightsTemp);

        free(bitmask);
        free(weightsUp);
        free(weightsDown);
        free(weightsLeft);
        free(weightsRight);
        free(heights);
        free(excessFlows);

    }
}
